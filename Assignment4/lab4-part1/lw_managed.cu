
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>
#include <stdlib.h>
#include <string>

__global__ void myKernel(int *m, int *v, int *r){
    // write your code here
        // write your code here
    uint thread_global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("block[%d], thread[%d]: m[%d]=%d\n", blockIdx.x, threadIdx.x, thread_global_idx, m[thread_global_idx]);
    r[thread_global_idx] = 0;
    
    for(int i =0; i < 3;i++)
    {
        r[thread_global_idx] += m[(thread_global_idx*3)+i]*v[i];
    }  
}

int main(int argc, char* argv[]){

    uint size_m = 9*sizeof(int);
    uint size_v = 3*sizeof(int);
    int *m, *v, *r;
    hipMallocManaged(&m, size_m);
    hipMallocManaged(&v, size_v);
    hipMallocManaged(&r, size_v);
    
    m[0]=0; m[1]=1; m[2]=2;
    m[3]=2; m[4]=3; m[5]=4;
    m[6]=4; m[7]=5; m[8]=6;

    v[0]=-1; v[1]=0; v[2]=1;

    uint b = 1; // dim3 b(1,1,1);
    uint t = 3; //dim3 t(3,1,1);
    myKernel<<<b,t>>>(m, v, r);
    hipDeviceSynchronize();
    
    int scs = 1;
    for(uint i=0; i<3; i++){
        if(r[i] != 2){
            printf("error! ");
            scs = 0;
        }
        printf("r[%d] = %d\n", i, r[i]);
    }
    if(scs == 1){
        printf("Done with no error\n");
    }
    hipFree(m);
    hipFree(v);
    hipFree(r);
    return 0;
}

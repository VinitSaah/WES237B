#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <assert.h>

#include "linear_layer.h"
#include "nn_exception.h"

using namespace std;
using namespace cv;

__global__ void linearLayerForward(float *W, float* input, float* output, float* b,
									const int W_rows, const int W_cols,
									const int input_rows, const int input_cols) 
{
    //TODO: complete the linear layer forward propagation
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    //Get dimension of the output 
    int output_rows = W_rows;
    int output_cols = input_cols;

    float output_value = 0.0f;

    if (col < output_cols && row < output_rows)
    {
        for (int i = 0; i < W_cols; i++)
        {
            output_value += W[row * W_cols + i] * input[i * input_cols + col];
        }

        output[row * output_cols + col] = output_value + b[row];
    }
}

__global__ void linearLayerBackprop(float *W, float* eB, float* eA,
									const int W_rows, const int W_cols,
									const int eB_rows, const int eB_cols) 
{
    //TODO: complete the linear layer backpropagation
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    //outputet dimension of the output. W is treated as transposed
    int eA_rows = W_cols;
    int eA_cols = eB_cols;

    float eA_value = 0.0f;

    if (row < eA_rows && col < eA_cols)
    {
        for (int i = 0; i < W_rows; i++)
        {
            eA_value += W[i * W_cols + row] * eB[i * eB_cols + col];
        }
        eA[row * eA_cols + col] = eA_value;
    }
}

__global__ void linearLayerUpdateWeights(float *eB, float* input, float* W,
									const int eB_rows, const int eB_cols,
									const int input_rows, const int input_cols, float learning_rate)
{
    //TODO: complete the gradient descent for weight updates
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    //outputet dimension of the output. input is treated as transposed
    int W_rows = eB_rows;
    int W_cols = input_rows;


    float dW_value = 0.0f;

    if (row < W_rows && col < W_cols)
    {
        for (int i = 0; i < eB_cols; i++)
        {
            dW_value += eB[row * eB_cols + i] * input[col * input_cols + i];
        }
        W[row * W_cols + col] = W[row * W_cols + col] - learning_rate * (dW_value / input_rows);
    }

}

__global__ void linearLayerUpdateBias(float *eB, float* b,
									const int eB_rows, const int eB_cols,
									const int b_rows, float learning_rate)
{
    //TODO: complete the gradient descent for bias updates
	int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < eB_rows * eB_cols)
    {
        int col = index % eB_cols;
        int row = index / eB_cols;
        atomicAdd(&b[row], -learning_rate * (eB[row * eB_cols + col] / eB_cols));
    }

}

LinearLayer::LinearLayer(string name, Shape W_shape) 
{
	W_shape.transpose();
	
	Matrix weights(W_shape);
	Matrix bias(W_shape.rows, 1);

	this->W = weights;
	this->b = bias;

	this->name = name;
	b.allocateMemory();
	W.allocateMemory();
	initializeBiasWithZeros();
	initializeWeightsRandomly();
}

LinearLayer::~LinearLayer() {}

void LinearLayer::initializeWeightsRandomly() 
{
	
	float mean = 0.0;
	float stddev = 1.0;

	theRNG().state = time(NULL);
	randn(W.data_host, Scalar(mean), Scalar(stddev));

	W.copyHostToDevice();
}

void LinearLayer::initializeWeightsHalf() 
{
	W.data_host = Scalar(0.5f);

	W.copyHostToDevice();
}


void LinearLayer::initializeBiasWithZeros()
{
	
	b.data_host = Scalar(0.0f);

	b.copyHostToDevice();
}

Matrix& LinearLayer::forward(Matrix& input)
{
//	printf("W shape : (%lu %lu)\n", W.shape.rows, W.shape.cols);
//	printf("input shape : (%lu %lu)\n", input.shape.rows, input.shape.cols);

	assert(W.shape.cols == input.shape.rows);
	
	this->input = input;

	Shape output_shape(W.shape.rows, input.shape.cols);

	output.allocateMemoryIfNotAllocated(output_shape);

	computeAndStoreLayerOutput(input);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform linear layer forward propagation");

	return output;
}

void LinearLayer::computeAndStoreLayerOutput(Matrix& input) {
	dim3 block_size(8, 8);
	dim3 num_of_blocks(	(output.shape.cols + block_size.x - 1) / block_size.x,
						(output.shape.rows + block_size.y - 1) / block_size.y);

	linearLayerForward<<<num_of_blocks, block_size>>>( W.data_device,
															input.data_device,
															output.data_device,
															b.data_device,
															W.shape.rows, W.shape.cols,
															input.shape.rows, input.shape.cols);
}

Matrix& LinearLayer::backprop(Matrix& eB, float learning_rate)
{
	eA.allocateMemoryIfNotAllocated(input.shape);

	computeAndStoreBackpropError(eB);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform back propagation.");

	updateBias(eB, learning_rate);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform bias update.");

	updateWeights(eB, learning_rate);
	NNException::throwIfDeviceErrorsOccurred("Cannot perform weights update.");

	return eA;
}


void LinearLayer::computeAndStoreBackpropError(Matrix& eB) {
	dim3 block_size(8, 8);
	dim3 num_of_blocks(	(input.shape.cols + block_size.x - 1) / block_size.x,
						(input.shape.rows + block_size.y - 1) / block_size.y);

	linearLayerBackprop<<<num_of_blocks, block_size>>>( W.data_device,
															eB.data_device,
															eA.data_device,
															W.shape.rows, W.shape.cols,
															eB.shape.rows, eB.shape.cols);
}

void LinearLayer::updateWeights(Matrix& eB, float learning_rate) {
	dim3 block_size(8, 8);
	dim3 num_of_blocks(	(W.shape.cols + block_size.x - 1) / block_size.x,
						(W.shape.rows + block_size.y - 1) / block_size.y);

	linearLayerUpdateWeights<<<num_of_blocks, block_size>>>(eB.data_device,
															input.data_device,
															W.data_device,
															eB.shape.rows, eB.shape.cols,
															input.shape.rows, input.shape.cols,
															learning_rate);
}

void LinearLayer::updateBias(Matrix& eB, float learning_rate) {
	dim3 block_size(256);
	dim3 num_of_blocks( (eB.shape.rows * eB.shape.cols + block_size.x - 1) / block_size.x);

	linearLayerUpdateBias<<<num_of_blocks, block_size>>>(eB.data_device,
															b.data_device,
															eB.shape.rows, eB.shape.cols,
															b.shape.rows, learning_rate);
}

int LinearLayer::getXDim() const {
	return W.shape.cols;
}

int LinearLayer::getYDim() const {
	return W.shape.rows;
}

Matrix LinearLayer::getWeightsMatrix() const {
	return W;
}

Matrix LinearLayer::getBiasVector() const {
	return b;
}
